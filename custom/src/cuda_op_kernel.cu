#define EIGEN_USE_GPU
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void AddOneKernel(const int* in, const int N, int* out) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
       i += blockDim.x * gridDim.x) {
    out[i] = in[i] + 1;
  }
}

void AddOneKernelLauncher(const int* in, const int N, int* out) {
  AddOneKernel<<<32, 256>>>(in, N, out);

  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess)
    printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
}